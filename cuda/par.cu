#include "hip/hip_runtime.h"
#include <vector>
#include <iostream>
#include <chrono>
#include <cmath>
#include <numeric>
//#include "argparse.hpp"
#include <cassert>
#include <fstream>
typedef float real_t;
#include <hip/hip_cooperative_groups.h>
using namespace cooperative_groups;


__constant__ size_t N_x;
__constant__ size_t N_y;
__constant__ real_t d_x;
__constant__ real_t d_y;
__constant__ real_t d_t;
__constant__ real_t C_i;
__constant__ real_t K_i;
__constant__ real_t D_p_i;
__constant__ real_t D_q_i;

class Sim_Configuration {
public:
    int iter = 1000;  // Number of iterations
    int data_period = 100;  // how often to save coordinate to file
    int size = 1024;
    real_t K = 9.0, C = 4.5, D_p = 1.0, D_q = 8.0;
    std::string filename = "chemicals.data";   // name of the output file with history

    Sim_Configuration(std::vector <std::string> argument){
        for (long unsigned int i = 1; i<argument.size() ; i += 2){
            std::string arg = argument[i];
            if(arg=="-h"){ // Write help
                std::cout << "./par --iter <number of iterations> --dt <time step>"
                          << " --g <gravitational const> --dx <x grid size> --dy <y grid size>"
                          << "--fperiod <iterations between each save> --out <name of output file>\n";
                exit(0);
            } else if (i == argument.size() - 1) {
                throw std::invalid_argument("The last argument (" + arg +") must have a value");
            } else if(arg=="--iter"){
                if ((iter = std::stoi(argument[i+1])) < 0) 
                    throw std::invalid_argument("iter most be a positive integer (e.g. -iter 1000)");
            } else if (arg=="--size"){
                if ((size = std::stoi(argument[i+1])) < 0) 
                    throw std::invalid_argument("size most be a positive integer (e.g. --size 100)");
            } else if(arg=="--fperiod"){
                if ((data_period = std::stoi(argument[i+1])) < 0) 
                    throw std::invalid_argument("dy most be a positive integer (e.g. -fperiod 100)");

            } else if(arg=="--out"){
                filename = argument[i+1];
            } else{
                std::cout << "---> error: the argument type is not recognized \n";
            }
        }
    }
};

/** Representation of the Chemicals domain including ghost lines, which is a "1-cell padding" of rows and columns
 *  around the world. These ghost lines are used to implement Neumann Boundary conditions. */
class Chemicals {
public:
    size_t NX, NY; // The shape of the Chemicals world including ghost lines.
    real_t D_p, D_q, C, K; 
    real_t dx, dy, dt;
    std::vector<real_t> P; // Virtual Chemical 1;
    std::vector<real_t> Q; // Virtual Chemical 2;
    std::vector<real_t> dPdt; // Time derivative of Chemical 1;
    std::vector<real_t> dQdt; // Time derivative of Chemical 2;

    Chemicals(size_t NX, size_t NY, real_t K = 9.0, real_t C = 4.5, real_t D_p = 1.0, real_t D_q = 8.0) : 
        P(NX*NY, 0), Q(NX*NY, 0), dQdt(NX*NY,0), dPdt(NX*NY,0), NX(NX), NY(NY), K(K), C(C), D_p(D_p), D_q(D_q)
    {
        //Initial conditions is a centered square of different initial concentrations.
        for (size_t i = NY/4; i < NY - NY/4 ; ++i) 
        for (size_t j = NX/4; j < NX - NX/4 ; ++j) {
            Q[i*NX + j] =  (K / C) + 0.2;
            P[i*NX + j] =  C + 0.1;
        }

        //Largest approximate timestep determined empirically from function fitting.
        dt = 0.02 * std::pow(real_t(std::min(NX,NY))/40.0,-2);

        //The effective spatial domain is kept fixed by adjusting the spatial resolution dx, dy.
        dx = 40/real_t(NX); dy = 40.0/real_t(NY);
    }
};

void to_file(const std::vector<std::vector<real_t>> &chemical_history, const std::string &filename){
    std::ofstream file(filename);
    std::cout << "Produced " << chemical_history.size() << " frames" <<std::endl;
    for (size_t i = 0; i < chemical_history.size(); i++)
    {
        file.write((const char*)(chemical_history.at(i).data()), sizeof(real_t)*chemical_history.at(0).size());
    }
    
}

/** Exchange the horizontal ghost lines i.e. copy the second data row to the very last data row and vice versa.
 *
 * @param data   The data update, which could be the Chemicals P or Q.
 * @param shape  The shape of data including the ghost lines.
 */
 
 __global__
void horizontal_boundary(real_t* data) {        
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N_x; i += stride){
        const uint64_t top_inside = 2 * N_x + i;
        const uint64_t bot_ghost = (N_y - 1) * N_x + i;
        const uint64_t bot_inside = (N_y - 3) * N_x + i;
        const uint64_t top_ghost = 0 * N_x + i;

        data[bot_ghost]  = data[bot_inside];
        data[top_ghost]  = data[top_inside];   
    }
}

/** Exchange the vertical ghost lines.
 *
 * @param data   The data update, which could be the Chemicals P or Q.
 * @param shape  The shape of data including the ghost lines.
 */
 
 __global__
void vertical_boundary(real_t* data) {
    
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N_y; i += stride){
    
        const uint64_t left_ghost  = i * N_y + 0;
        const uint64_t right_inside = i * N_y + N_x - 3;
        const uint64_t right_ghost = i * N_x + N_x - 1;
        const uint64_t left_inside  = i * N_x + 2;
        
        data[left_ghost]  = data[right_inside];
        data[right_ghost] = data[left_inside];   
    }
}



/** One integration step
 *
 * @param c The Chemicals world to update.
 */
__global__
void integrate(real_t* P, real_t* Q, real_t* dPdt, real_t* dQdt) {

    int index = blockIdx.x * blockDim.x + threadIdx.x + N_x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < N_y*N_x - N_x; i += stride){
        real_t P2Q = P[i]*P[i] * Q[i];
        dPdt[i] = D_p_i * ( P[i - 1] + P[i + 1] + P[i+N_x] + P[i-N_x] - 4*P[i]) / (d_x * d_y) + C_i + P2Q - (K_i + 1.0) * P[i];
        dQdt[i] = D_q_i * ( Q[i - 1] + Q[i + 1] + Q[i+N_x] + Q[i-N_x] - 4*Q[i]) / (d_x * d_y) + P[i] * K_i - P2Q;
    
    }
    
    index = blockIdx.x * blockDim.x + threadIdx.x + N_x;
    stride = blockDim.x * gridDim.x;
    for (int i = index; i < N_y*N_x - N_x; i += stride){
            //Euler integration
            P[i] += d_t * dPdt[i];
            Q[i] += d_t * dQdt[i];
        }
}

/** Simulation of Chemicals
 *
 * @param num_of_iterations  The number of time steps to simulate
 * @param size               The x and y domain size
 * @param output_filename    The filename of the written Chemicals history
 */
void simulate(const Sim_Configuration &config) {
    using namespace std::chrono_literals;
    // We pad the world with ghost lines (two in each dimension)
    Chemicals chemicals = Chemicals(config.size, config.size);

    std::vector <std::vector<real_t>> chemicals_history;
    double checksum = 0;
    size_t N = chemicals.NX * chemicals.NY;
    
    real_t* P = chemicals.P.data(); 
    real_t* Q = chemicals.Q.data();

    hipMemcpyToSymbol(HIP_SYMBOL(N_x), &chemicals.NX, sizeof(size_t));
    hipMemcpyToSymbol(HIP_SYMBOL(N_y), &chemicals.NY, sizeof(size_t));
    hipMemcpyToSymbol(HIP_SYMBOL(d_x), &chemicals.dx, sizeof(real_t));
    hipMemcpyToSymbol(HIP_SYMBOL(d_y), &chemicals.dy, sizeof(real_t));
    hipMemcpyToSymbol(HIP_SYMBOL(d_t), &chemicals.dt, sizeof(real_t));
    hipMemcpyToSymbol(HIP_SYMBOL(D_p_i), &chemicals.D_p, sizeof(real_t));
    hipMemcpyToSymbol(HIP_SYMBOL(D_q_i), &chemicals.D_q, sizeof(real_t));
    hipMemcpyToSymbol(HIP_SYMBOL(C_i), &chemicals.C, sizeof(real_t));
    hipMemcpyToSymbol(HIP_SYMBOL(K_i), &chemicals.K, sizeof(real_t));
    
    real_t *p;
    hipMalloc((void **)&p, N*sizeof(real_t));
    hipMemcpy(p, chemicals.P.data(), N*sizeof(real_t), hipMemcpyHostToDevice);
        
    real_t *q;
    hipMalloc((void **)&q, N*sizeof(real_t));
    hipMemcpy(q, chemicals.Q.data(), N*sizeof(real_t), hipMemcpyHostToDevice);
    
    real_t *dp;
    hipMalloc((void **)&dp, N*sizeof(real_t));
    hipMemcpy(dp, chemicals.dPdt.data(), N*sizeof(real_t), hipMemcpyHostToDevice);
    
    real_t *dq;
    hipMalloc((void **)&dq, N*sizeof(real_t));
    hipMemcpy(dq, chemicals.dQdt.data(), N*sizeof(real_t), hipMemcpyHostToDevice);
    
    int threads = 1024;
    int blocks = 14;
    
    auto begin = std::chrono::steady_clock::now();
    
    for (uint64_t t = 0; t < config.iter; ++t) {
        
        vertical_boundary<<<blocks,threads>>>(p);
        vertical_boundary<<<blocks,threads>>>(q);
        horizontal_boundary<<<blocks,threads>>>(p);
        horizontal_boundary<<<blocks,threads>>>(q);
        
        integrate<<<blocks,threads>>>(p,q,dp,dq);
        
        if (t % config.data_period == 0) {
            hipMemcpy(chemicals.P.data(), p, N*sizeof(real_t), hipMemcpyDeviceToHost);
            chemicals_history.push_back(chemicals.P);
            std::cout << t;
            
        }
    
    }
    hipDeviceSynchronize();
    auto end = std::chrono::steady_clock::now();

    /**If you want to check the output: **/
    to_file(chemicals_history, config.filename);    
    
    hipMemcpy(chemicals.P.data(), p, N*sizeof(real_t), hipMemcpyDeviceToHost);
    checksum += std::accumulate(chemicals.P.begin(), chemicals.P.end(), 0.0);
    std::cout << "checksum: " << checksum << std::endl;
    std::cout << "elapsed time: " << (end - begin).count() / 1000000000.0 << " sec" << std::endl;
}

/** Main function that parses the command line and start the simulation */
int main(int argc, char **argv) {
    auto config = Sim_Configuration({argv, argv+argc});
    simulate(config);
    return 0;
}
__global__ void device_copy_vector4_kernel(int* d_in, int* d_out, int N) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = idx; i < N/4; i += blockDim.x * gridDim.x) {
    reinterpret_cast<real_t*>(d_out)[i] = reinterpret_cast<int4*>(d_in)[i];
  }